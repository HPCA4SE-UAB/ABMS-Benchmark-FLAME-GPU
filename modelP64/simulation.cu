#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.4.0 for CUDA 6
 * Copyright 2015 University of Sheffield.
 * Author: Dr Paul Richmond 
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence 
 * on www.flamegpu.com website.
 * 
 */

//Disable internal thrust warnings about conversions
#pragma warning(push)
#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

// includes
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#include <vector_operators.h>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"

#pragma warning(pop)

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
   
}

/* SM padding and offset variables */
int SM_START;
int PADDING;

/* Agent Memory */

/* person Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_person_list* d_persons;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_person_list* d_persons_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_person_list* d_persons_new;  /**< Pointer to new agent list on the device (used to hold new agents bfore they are appended to the population)*/
int h_xmachine_memory_person_count;   /**< Agent population size counter */ 
uint * d_xmachine_memory_person_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_person_values;  /**< Agent sort identifiers value */
    
/* person state variables */
xmachine_memory_person_list* h_persons_start;      /**< Pointer to agent list (population) on host*/
xmachine_memory_person_list* d_persons_start;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_person_start_count;   /**< Agent population size counter */
int agent_number;                           /* Agent number read from 0.xml file */ 

/* person state variables */
xmachine_memory_person_list* h_persons_1;      /**< Pointer to agent list (population) on host*/
xmachine_memory_person_list* d_persons_1;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_person_1_count;   /**< Agent population size counter */ 

/* person state variables */
xmachine_memory_person_list* h_persons_2;      /**< Pointer to agent list (population) on host*/
xmachine_memory_person_list* d_persons_2;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_person_2_count;   /**< Agent population size counter */ 

/* person state variables */
xmachine_memory_person_list* h_persons_3;      /**< Pointer to agent list (population) on host*/
xmachine_memory_person_list* d_persons_3;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_person_3_count;   /**< Agent population size counter */ 


/* Message Memory */

/* agentLocation Message variables */
xmachine_message_agentLocation_list* h_agentLocations;         /**< Pointer to message list on host*/
xmachine_message_agentLocation_list* d_agentLocations;         /**< Pointer to message list on device*/
xmachine_message_agentLocation_list* d_agentLocations_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Non partitioned and spatial partitioned message variables  */
int h_message_agentLocation_count;         /**< message list counter*/
int h_message_agentLocation_output_type;   /**< message output type (single or optional)*/
/* Spatial Partitioning Variables*/
#ifdef FAST_ATOMIC_SORTING
	uint * d_xmachine_message_agentLocation_local_bin_index;	  /**< index offset within the assigned bin */
	uint * d_xmachine_message_agentLocation_unsorted_index;		/**< unsorted index (hash) value for message */
#else
	uint * d_xmachine_message_agentLocation_keys;	  /**< message sort identifier keys*/
	uint * d_xmachine_message_agentLocation_values;  /**< message sort identifier values */
#endif
xmachine_message_agentLocation_PBM * d_agentLocation_partition_matrix;  /**< Pointer to PCB matrix */
float3 h_message_agentLocation_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
float3 h_message_agentLocation_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
int3 h_message_agentLocation_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
float h_message_agentLocation_radius;                 /**< partition radius (used to determin the size of the partitions) */
/* Texture offset values for host */
int h_tex_xmachine_message_agentLocation_my_id_offset;
int h_tex_xmachine_message_agentLocation_x_offset;
int h_tex_xmachine_message_agentLocation_y_offset;
int h_tex_xmachine_message_agentLocation_z_offset;
int h_tex_xmachine_message_agentLocation_pbm_start_offset;
int h_tex_xmachine_message_agentLocation_pbm_end_or_count_offset;

/* agentCooperate Message variables */
xmachine_message_agentCooperate_list* h_agentCooperates;         /**< Pointer to message list on host*/
xmachine_message_agentCooperate_list* d_agentCooperates;         /**< Pointer to message list on device*/
xmachine_message_agentCooperate_list* d_agentCooperates_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Non partitioned and spatial partitioned message variables  */
int h_message_agentCooperate_count;         /**< message list counter*/
int h_message_agentCooperate_output_type;   /**< message output type (single or optional)*/
/* Spatial Partitioning Variables*/
#ifdef FAST_ATOMIC_SORTING
	uint * d_xmachine_message_agentCooperate_local_bin_index;	  /**< index offset within the assigned bin */
	uint * d_xmachine_message_agentCooperate_unsorted_index;		/**< unsorted index (hash) value for message */
#else
	uint * d_xmachine_message_agentCooperate_keys;	  /**< message sort identifier keys*/
	uint * d_xmachine_message_agentCooperate_values;  /**< message sort identifier values */
#endif
xmachine_message_agentCooperate_PBM * d_agentCooperate_partition_matrix;  /**< Pointer to PCB matrix */
float3 h_message_agentCooperate_min_bounds;           /**< min bounds (x,y,z) of partitioning environment */
float3 h_message_agentCooperate_max_bounds;           /**< max bounds (x,y,z) of partitioning environment */
int3 h_message_agentCooperate_partitionDim;           /**< partition dimensions (x,y,z) of partitioning environment */
float h_message_agentCooperate_radius;                 /**< partition radius (used to determin the size of the partitions) */
/* Texture offset values for host */
int h_tex_xmachine_message_agentCooperate_source_id_offset;
int h_tex_xmachine_message_agentCooperate_destination_id_offset;
int h_tex_xmachine_message_agentCooperate_cooperate_offset;
int h_tex_xmachine_message_agentCooperate_x_offset;
int h_tex_xmachine_message_agentCooperate_y_offset;
int h_tex_xmachine_message_agentCooperate_z_offset;
int h_tex_xmachine_message_agentCooperate_message0_offset;
int h_tex_xmachine_message_agentCooperate_message1_offset;
int h_tex_xmachine_message_agentCooperate_message2_offset;
int h_tex_xmachine_message_agentCooperate_message3_offset;
int h_tex_xmachine_message_agentCooperate_message4_offset;
int h_tex_xmachine_message_agentCooperate_message5_offset;
int h_tex_xmachine_message_agentCooperate_message6_offset;
int h_tex_xmachine_message_agentCooperate_message7_offset;
int h_tex_xmachine_message_agentCooperate_message8_offset;
int h_tex_xmachine_message_agentCooperate_message9_offset;
int h_tex_xmachine_message_agentCooperate_message10_offset;
int h_tex_xmachine_message_agentCooperate_message11_offset;
int h_tex_xmachine_message_agentCooperate_message12_offset;
int h_tex_xmachine_message_agentCooperate_pbm_start_offset;
int h_tex_xmachine_message_agentCooperate_pbm_end_or_count_offset;

  
/* CUDA Streams for function layers */
hipStream_t stream1;

/*Global condition counts*/

/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* CUDA Parallel Primatives variables */
int scan_last_sum;           /**< Indicates if the position (in message list) of last message*/
int scan_last_included;      /**< Indicates if last sum value is included in the total sum count*/

/* Agent function prototypes */

/** person_walk
 * Agent function prototype for walk function of person agent
 */
void person_walk(hipStream_t &stream);

/** person_cooperate
 * Agent function prototype for cooperate function of person agent
 */
void person_cooperate(hipStream_t &stream);

/** person_play
 * Agent function prototype for play function of person agent
 */
void person_play(hipStream_t &stream);

/** person_compute
 * Agent function prototype for compute function of person agent
 */
void person_compute(hipStream_t &stream);

  
void setPaddingAndOffset()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(0);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
	printf("Simulation requires full precision double values\n");
	if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
		printf("Error: Hardware does not support full precision double values!\n");
		exit(0);
	}
    
#endif

	//check 32 or 64bit
	x64_sys = (sizeof(void*)==8);
	if (x64_sys)
	{
		printf("64Bit System Detected\n");
	}
	else
	{
		printf("32Bit System Detected\n");
	}

	SM_START = 0;
	PADDING = 0;
  
	//copy padding and offset to GPU
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));     
}

int closest_sqr_pow2(int x){
	int h, h_d;
	int l, l_d;
	
	//higher bound
	h = (int)pow(4, ceil(log(x)/log(4)));
	h_d = h-x;
	
	//escape early if x is square power of 2
	if (h_d == x)
		return x;
	
	//lower bound		
	l = (int)pow(4, floor(log(x)/log(4)));
	l_d = x-l;
	
	//closest bound
	if(h_d < l_d)
		return h;
	else 
		return l;
}

int is_sqr_pow2(int x){
	int r = (int)pow(4, ceil(log(x)/log(4)));
	return (r == x);
}

/* Unary function required for hipOccupancyMaxPotentialBlockSizeVariableSMem to avoid warnings */
int no_sm(int b){
	return 0;
}

/* Unary function to return shared memory size for reorder message kernels */
int reorder_messages_sm_size(int blockSize)
{
	return sizeof(unsigned int)*(blockSize+1);
}


void initialise(char * inputfile){

	//set the padding and offset values depending on architecture and OS
	setPaddingAndOffset();
  

	printf("Allocating Host and Device memeory\n");
  
	/* Agent memory allocation (CPU) */
	int xmachine_person_SoA_size = sizeof(xmachine_memory_person_list);
	h_persons_start = (xmachine_memory_person_list*)malloc(xmachine_person_SoA_size);
	h_persons_1 = (xmachine_memory_person_list*)malloc(xmachine_person_SoA_size);
	h_persons_2 = (xmachine_memory_person_list*)malloc(xmachine_person_SoA_size);
	h_persons_3 = (xmachine_memory_person_list*)malloc(xmachine_person_SoA_size);

	/* Message memory allocation (CPU) */
	int message_agentLocation_SoA_size = sizeof(xmachine_message_agentLocation_list);
	h_agentLocations = (xmachine_message_agentLocation_list*)malloc(message_agentLocation_SoA_size);
	int message_agentCooperate_SoA_size = sizeof(xmachine_message_agentCooperate_list);
	h_agentCooperates = (xmachine_message_agentCooperate_list*)malloc(message_agentCooperate_SoA_size);

	//Exit if agent or message buffer sizes are to small for function outpus
			
	/* Set spatial partitioning agentLocation message variables (min_bounds, max_bounds)*/
	h_message_agentLocation_radius = (float)10;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_radius), &h_message_agentLocation_radius, sizeof(float)));	
	    h_message_agentLocation_min_bounds = make_float3((float)0.0, (float)0.0, (float)0.0);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_min_bounds), &h_message_agentLocation_min_bounds, sizeof(float3)));	
	h_message_agentLocation_max_bounds = make_float3((float)300, (float)300, (float)300);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_max_bounds), &h_message_agentLocation_max_bounds, sizeof(float3)));	
	h_message_agentLocation_partitionDim.x = (int)ceil((h_message_agentLocation_max_bounds.x - h_message_agentLocation_min_bounds.x)/h_message_agentLocation_radius);
	h_message_agentLocation_partitionDim.y = (int)ceil((h_message_agentLocation_max_bounds.y - h_message_agentLocation_min_bounds.y)/h_message_agentLocation_radius);
	h_message_agentLocation_partitionDim.z = (int)ceil((h_message_agentLocation_max_bounds.z - h_message_agentLocation_min_bounds.z)/h_message_agentLocation_radius);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_partitionDim), &h_message_agentLocation_partitionDim, sizeof(int3)));	
	
			
	/* Set spatial partitioning agentCooperate message variables (min_bounds, max_bounds)*/
	h_message_agentCooperate_radius = (float)10;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_radius), &h_message_agentCooperate_radius, sizeof(float)));	
	    h_message_agentCooperate_min_bounds = make_float3((float)0.0, (float)0.0, (float)0.0);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_min_bounds), &h_message_agentCooperate_min_bounds, sizeof(float3)));	
	h_message_agentCooperate_max_bounds = make_float3((float)300, (float)300, (float)300);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_max_bounds), &h_message_agentCooperate_max_bounds, sizeof(float3)));	
	h_message_agentCooperate_partitionDim.x = (int)ceil((h_message_agentCooperate_max_bounds.x - h_message_agentCooperate_min_bounds.x)/h_message_agentCooperate_radius);
	h_message_agentCooperate_partitionDim.y = (int)ceil((h_message_agentCooperate_max_bounds.y - h_message_agentCooperate_min_bounds.y)/h_message_agentCooperate_radius);
	h_message_agentCooperate_partitionDim.z = (int)ceil((h_message_agentCooperate_max_bounds.z - h_message_agentCooperate_min_bounds.z)/h_message_agentCooperate_radius);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_partitionDim), &h_message_agentCooperate_partitionDim, sizeof(int3)));	
	

	//read initial states
	readInitialStates(inputfile, h_persons_start, &h_xmachine_memory_person_start_count);
	
	
	/* person Agent memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_persons, xmachine_person_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_persons_swap, xmachine_person_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_persons_new, xmachine_person_SoA_size));
    //continuous agent sort identifiers
  gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_person_keys, xmachine_memory_person_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_person_values, xmachine_memory_person_MAX* sizeof(uint)));
	/* start memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_persons_start, xmachine_person_SoA_size));
	gpuErrchk( hipMemcpy( d_persons_start, h_persons_start, xmachine_person_SoA_size, hipMemcpyHostToDevice));
    
	/* 1 memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_persons_1, xmachine_person_SoA_size));
	gpuErrchk( hipMemcpy( d_persons_1, h_persons_1, xmachine_person_SoA_size, hipMemcpyHostToDevice));
    
	/* 2 memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_persons_2, xmachine_person_SoA_size));
	gpuErrchk( hipMemcpy( d_persons_2, h_persons_2, xmachine_person_SoA_size, hipMemcpyHostToDevice));
    
	/* 3 memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_persons_3, xmachine_person_SoA_size));
	gpuErrchk( hipMemcpy( d_persons_3, h_persons_3, xmachine_person_SoA_size, hipMemcpyHostToDevice));
    
	/* agentLocation Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_agentLocations, message_agentLocation_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_agentLocations_swap, message_agentLocation_SoA_size));
	gpuErrchk( hipMemcpy( d_agentLocations, h_agentLocations, message_agentLocation_SoA_size, hipMemcpyHostToDevice));
	gpuErrchk( hipMalloc( (void**) &d_agentLocation_partition_matrix, sizeof(xmachine_message_agentLocation_PBM)));
#ifdef FAST_ATOMIC_SORTING
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentLocation_local_bin_index, xmachine_message_agentLocation_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentLocation_unsorted_index, xmachine_message_agentLocation_MAX* sizeof(uint)));
#else
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentLocation_keys, xmachine_message_agentLocation_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentLocation_values, xmachine_message_agentLocation_MAX* sizeof(uint)));
#endif
	
	/* agentCooperate Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_agentCooperates, message_agentCooperate_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_agentCooperates_swap, message_agentCooperate_SoA_size));
	gpuErrchk( hipMemcpy( d_agentCooperates, h_agentCooperates, message_agentCooperate_SoA_size, hipMemcpyHostToDevice));
	gpuErrchk( hipMalloc( (void**) &d_agentCooperate_partition_matrix, sizeof(xmachine_message_agentCooperate_PBM)));
#ifdef FAST_ATOMIC_SORTING
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentCooperate_local_bin_index, xmachine_message_agentCooperate_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentCooperate_unsorted_index, xmachine_message_agentCooperate_MAX* sizeof(uint)));
#else
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentCooperate_keys, xmachine_message_agentCooperate_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_message_agentCooperate_values, xmachine_message_agentCooperate_MAX* sizeof(uint)));
#endif
		

	/*Set global condition counts*/

	/* RNG rand48 */
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	gpuErrchk( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	gpuErrchk( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

	/* Call all init functions */
	initConstants();
	
  
  /* Init CUDA Streams for function layers */
  
  gpuErrchk(hipStreamCreate(&stream1));
} 


void sort_persons_start(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_person_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_person_start_count); 
	gridSize = (h_xmachine_memory_person_start_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_person_keys, d_xmachine_memory_person_values, d_persons_start);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_person_keys),  thrust::device_pointer_cast(d_xmachine_memory_person_keys) + h_xmachine_memory_person_start_count,  thrust::device_pointer_cast(d_xmachine_memory_person_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_person_agents, no_sm, h_xmachine_memory_person_start_count); 
	gridSize = (h_xmachine_memory_person_start_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_person_agents<<<gridSize, blockSize>>>(d_xmachine_memory_person_values, d_persons_start, d_persons_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_person_list* d_persons_temp = d_persons_start;
	d_persons_start = d_persons_swap;
	d_persons_swap = d_persons_temp;	
}

void sort_persons_1(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_person_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_person_1_count); 
	gridSize = (h_xmachine_memory_person_1_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_person_keys, d_xmachine_memory_person_values, d_persons_1);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_person_keys),  thrust::device_pointer_cast(d_xmachine_memory_person_keys) + h_xmachine_memory_person_1_count,  thrust::device_pointer_cast(d_xmachine_memory_person_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_person_agents, no_sm, h_xmachine_memory_person_1_count); 
	gridSize = (h_xmachine_memory_person_1_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_person_agents<<<gridSize, blockSize>>>(d_xmachine_memory_person_values, d_persons_1, d_persons_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_person_list* d_persons_temp = d_persons_1;
	d_persons_1 = d_persons_swap;
	d_persons_swap = d_persons_temp;	
}

void sort_persons_2(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_person_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_person_2_count); 
	gridSize = (h_xmachine_memory_person_2_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_person_keys, d_xmachine_memory_person_values, d_persons_2);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_person_keys),  thrust::device_pointer_cast(d_xmachine_memory_person_keys) + h_xmachine_memory_person_2_count,  thrust::device_pointer_cast(d_xmachine_memory_person_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_person_agents, no_sm, h_xmachine_memory_person_2_count); 
	gridSize = (h_xmachine_memory_person_2_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_person_agents<<<gridSize, blockSize>>>(d_xmachine_memory_person_values, d_persons_2, d_persons_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_person_list* d_persons_temp = d_persons_2;
	d_persons_2 = d_persons_swap;
	d_persons_swap = d_persons_temp;	
}

void sort_persons_3(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_person_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_person_3_count); 
	gridSize = (h_xmachine_memory_person_3_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_person_keys, d_xmachine_memory_person_values, d_persons_3);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_person_keys),  thrust::device_pointer_cast(d_xmachine_memory_person_keys) + h_xmachine_memory_person_3_count,  thrust::device_pointer_cast(d_xmachine_memory_person_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_person_agents, no_sm, h_xmachine_memory_person_3_count); 
	gridSize = (h_xmachine_memory_person_3_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_person_agents<<<gridSize, blockSize>>>(d_xmachine_memory_person_values, d_persons_3, d_persons_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_person_list* d_persons_temp = d_persons_3;
	d_persons_3 = d_persons_swap;
	d_persons_swap = d_persons_temp;	
}


void cleanup(){

	/* Agent data free*/
	
	/* person Agent variables */
	gpuErrchk(hipFree(d_persons));
	gpuErrchk(hipFree(d_persons_swap));
	gpuErrchk(hipFree(d_persons_new));
	
	free( h_persons_start);
	gpuErrchk(hipFree(d_persons_start));
	
	free( h_persons_1);
	gpuErrchk(hipFree(d_persons_1));
	
	free( h_persons_2);
	gpuErrchk(hipFree(d_persons_2));
	
	free( h_persons_3);
	gpuErrchk(hipFree(d_persons_3));
	

	/* Message data free */
	
	/* agentLocation Message variables */
	free( h_agentLocations);
	gpuErrchk(hipFree(d_agentLocations));
	gpuErrchk(hipFree(d_agentLocations_swap));
	gpuErrchk(hipFree(d_agentLocation_partition_matrix));
#ifdef FAST_ATOMIC_SORTING
	gpuErrchk(hipFree(d_xmachine_message_agentLocation_local_bin_index));
	gpuErrchk(hipFree(d_xmachine_message_agentLocation_unsorted_index));
#else
	gpuErrchk(hipFree(d_xmachine_message_agentLocation_keys));
	gpuErrchk(hipFree(d_xmachine_message_agentLocation_values));
#endif
	
	/* agentCooperate Message variables */
	free( h_agentCooperates);
	gpuErrchk(hipFree(d_agentCooperates));
	gpuErrchk(hipFree(d_agentCooperates_swap));
	gpuErrchk(hipFree(d_agentCooperate_partition_matrix));
#ifdef FAST_ATOMIC_SORTING
	gpuErrchk(hipFree(d_xmachine_message_agentCooperate_local_bin_index));
	gpuErrchk(hipFree(d_xmachine_message_agentCooperate_unsorted_index));
#else
	gpuErrchk(hipFree(d_xmachine_message_agentCooperate_keys));
	gpuErrchk(hipFree(d_xmachine_message_agentCooperate_values));
#endif
	
  
  /* CUDA Streams for function layers */
  
  gpuErrchk(hipStreamDestroy(stream1));
}

void singleIteration(){

	/* set all non partitioned and spatial partitionded message counts to 0*/
	h_message_agentLocation_count = 0;
	//upload to device constant
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_count), &h_message_agentLocation_count, sizeof(int)));
	
	h_message_agentCooperate_count = 0;
	//upload to device constant
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_count), &h_message_agentCooperate_count, sizeof(int)));
	

	/* Call agent functions in order itterating through the layer functions */

        // Start profiling, JJRG 03-07-2018
        //hipProfilerStart ();	

	/* Layer 1*/
	person_walk(stream1);
	hipDeviceSynchronize();
  
	/* Layer 2*/
	person_cooperate(stream1);
	hipDeviceSynchronize();
  
	/* Layer 3*/
	person_play(stream1);
	hipDeviceSynchronize();
        // Stop profiling, JJRG 03-07-2018
        //hipProfilerStop ();
  
	/* Layer 4*/
	person_compute(stream1);
	hipDeviceSynchronize();
  
}

/* Environment functions */


void set_height(int* h_height){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(height), h_height, sizeof(int)));
}

void set_width(int* h_width){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(width), h_width, sizeof(int)));
}

void set_radius(int* h_radius){
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(radius), h_radius, sizeof(int)));
}


/* Agent data access functions*/

    
int get_agent_person_MAX_count(){
    return xmachine_memory_person_MAX;
}


int get_agent_person_start_count(){
	//continuous agent
	return h_xmachine_memory_person_start_count;
	
}

xmachine_memory_person_list* get_device_person_start_agents(){
	return d_persons_start;
}

xmachine_memory_person_list* get_host_person_start_agents(){
	return h_persons_start;
}

int get_agent_person_1_count(){
	//continuous agent
	return h_xmachine_memory_person_1_count;
	
}

xmachine_memory_person_list* get_device_person_1_agents(){
	return d_persons_1;
}

xmachine_memory_person_list* get_host_person_1_agents(){
	return h_persons_1;
}

int get_agent_person_2_count(){
	//continuous agent
	return h_xmachine_memory_person_2_count;
	
}

xmachine_memory_person_list* get_device_person_2_agents(){
	return d_persons_2;
}

xmachine_memory_person_list* get_host_person_2_agents(){
	return h_persons_2;
}

int get_agent_person_3_count(){
	//continuous agent
	return h_xmachine_memory_person_3_count;
	
}

xmachine_memory_person_list* get_device_person_3_agents(){
	return d_persons_3;
}

xmachine_memory_person_list* get_host_person_3_agents(){
	return h_persons_3;
}


/* Agent functions */


	
/* Shared memory size calculator for agent function */
int person_walk_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** person_walk
 * Agent function prototype for walk function of person agent
 */
void person_walk(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_person_start_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_person_start_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_person_list* persons_start_temp = d_persons;
	d_persons = d_persons_start;
	d_persons_start = persons_start_temp;
	//set working count to current state count
	h_xmachine_memory_person_count = h_xmachine_memory_person_start_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_count), &h_xmachine_memory_person_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_person_start_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_start_count), &h_xmachine_memory_person_start_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_agentLocation_count + h_xmachine_memory_person_count > xmachine_message_agentLocation_MAX){
		printf("Error: Buffer size of agentLocation message will be exceeded in function walk\n");
		exit(0);
	}
	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_walk, person_walk_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = person_walk_sm_size(blockSize);
	
	
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	//Set the message_type for non partitioned and spatially partitioned message outputs
	h_message_agentLocation_output_type = single_message;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_output_type), &h_message_agentLocation_output_type, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (walk)
	//Reallocate   : false
	//Input        : 
	//Output       : agentLocation
	//Agent Output : 
	GPUFLAME_walk<<<g, b, sm_size, stream>>>(d_persons, d_agentLocations, d_rand48);
	gpuErrchkLaunch();
	
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	h_message_agentLocation_count += h_xmachine_memory_person_count;	
	//Copy count to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentLocation_count), &h_message_agentLocation_count, sizeof(int)));	
	
#ifdef FAST_ATOMIC_SORTING
  //USE ATOMICS TO BUILD PARTITION BOUNDARY
	//reset partition matrix
	gpuErrchk( hipMemset( (void*) d_agentLocation_partition_matrix, 0, sizeof(xmachine_message_agentLocation_PBM)));
  //
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, hist_agentLocation_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	hist_agentLocation_messages<<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_agentLocation_local_bin_index, d_xmachine_message_agentLocation_unsorted_index, d_agentLocation_partition_matrix->end_or_count, d_agentLocations, state_list_size);
	gpuErrchkLaunch();
	
	thrust::device_ptr<int> ptr_count = thrust::device_pointer_cast(d_agentLocation_partition_matrix->end_or_count);
	thrust::device_ptr<int> ptr_index = thrust::device_pointer_cast(d_agentLocation_partition_matrix->start);
	thrust::exclusive_scan(thrust::cuda::par.on(stream), ptr_count, ptr_count + xmachine_message_agentLocation_grid_size, ptr_index); // scan
	
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_agentLocation_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize; 	// Round up according to array size 
	reorder_agentLocation_messages <<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_agentLocation_local_bin_index, d_xmachine_message_agentLocation_unsorted_index, d_agentLocation_partition_matrix->start, d_agentLocations, d_agentLocations_swap, state_list_size);
	gpuErrchkLaunch();
#else
	//HASH, SORT, REORDER AND BUILD PMB FOR SPATIAL PARTITIONING MESSAGE OUTPUTS
	//Get message hash values for sorting
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, hash_agentLocation_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	hash_agentLocation_messages<<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_agentLocation_keys, d_xmachine_message_agentLocation_values, d_agentLocations);
	gpuErrchkLaunch();
	//Sort
	thrust::sort_by_key(thrust::cuda::par.on(stream), thrust::device_pointer_cast(d_xmachine_message_agentLocation_keys),  thrust::device_pointer_cast(d_xmachine_message_agentLocation_keys) + h_message_agentLocation_count,  thrust::device_pointer_cast(d_xmachine_message_agentLocation_values));
	gpuErrchkLaunch();
	//reorder and build pcb
	gpuErrchk(hipMemset(d_agentLocation_partition_matrix->start, 0xffffffff, xmachine_message_agentLocation_grid_size* sizeof(int)));
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_agentLocation_messages, reorder_messages_sm_size, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	int reorder_sm_size = reorder_messages_sm_size(blockSize);
	reorder_agentLocation_messages<<<gridSize, blockSize, reorder_sm_size, stream>>>(d_xmachine_message_agentLocation_keys, d_xmachine_message_agentLocation_values, d_agentLocation_partition_matrix, d_agentLocations, d_agentLocations_swap);
	gpuErrchkLaunch();
#endif
	//swap ordered list
	xmachine_message_agentLocation_list* d_agentLocations_temp = d_agentLocations;
	d_agentLocations = d_agentLocations_swap;
	d_agentLocations_swap = d_agentLocations_temp;
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_person_1_count+h_xmachine_memory_person_count > xmachine_memory_person_MAX){
		printf("Error: Buffer size of walk agents in state 1 will be exceeded moving working agents to next state in function walk\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_person_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_person_Agents<<<gridSize, blockSize, 0, stream>>>(d_persons_1, d_persons, h_xmachine_memory_person_1_count, h_xmachine_memory_person_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_person_1_count += h_xmachine_memory_person_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_1_count), &h_xmachine_memory_person_1_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int person_cooperate_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Continuous agent and message input is spatially partitioned
	sm_size += (blockSize * sizeof(xmachine_message_agentLocation));
	
	//all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (blockSize * PADDING);
	
	return sm_size;
}

/** person_cooperate
 * Agent function prototype for cooperate function of person agent
 */
void person_cooperate(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_person_1_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_person_1_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_person_list* persons_1_temp = d_persons;
	d_persons = d_persons_1;
	d_persons_1 = persons_1_temp;
	//set working count to current state count
	h_xmachine_memory_person_count = h_xmachine_memory_person_1_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_count), &h_xmachine_memory_person_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_person_1_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_1_count), &h_xmachine_memory_person_1_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_agentCooperate_count + h_xmachine_memory_person_count > xmachine_message_agentCooperate_MAX){
		printf("Error: Buffer size of agentCooperate message will be exceeded in function cooperate\n");
		exit(0);
	}
	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_cooperate, person_cooperate_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = person_cooperate_sm_size(blockSize);
	
	
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//continuous agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_agentLocation_my_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentLocation_my_id_byte_offset, tex_xmachine_message_agentLocation_my_id, d_agentLocations->my_id, sizeof(int)*xmachine_message_agentLocation_MAX));
	h_tex_xmachine_message_agentLocation_my_id_offset = (int)tex_xmachine_message_agentLocation_my_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentLocation_my_id_offset), &h_tex_xmachine_message_agentLocation_my_id_offset, sizeof(int)));
	size_t tex_xmachine_message_agentLocation_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentLocation_x_byte_offset, tex_xmachine_message_agentLocation_x, d_agentLocations->x, sizeof(int)*xmachine_message_agentLocation_MAX));
	h_tex_xmachine_message_agentLocation_x_offset = (int)tex_xmachine_message_agentLocation_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentLocation_x_offset), &h_tex_xmachine_message_agentLocation_x_offset, sizeof(int)));
	size_t tex_xmachine_message_agentLocation_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentLocation_y_byte_offset, tex_xmachine_message_agentLocation_y, d_agentLocations->y, sizeof(int)*xmachine_message_agentLocation_MAX));
	h_tex_xmachine_message_agentLocation_y_offset = (int)tex_xmachine_message_agentLocation_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentLocation_y_offset), &h_tex_xmachine_message_agentLocation_y_offset, sizeof(int)));
	size_t tex_xmachine_message_agentLocation_z_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentLocation_z_byte_offset, tex_xmachine_message_agentLocation_z, d_agentLocations->z, sizeof(int)*xmachine_message_agentLocation_MAX));
	h_tex_xmachine_message_agentLocation_z_offset = (int)tex_xmachine_message_agentLocation_z_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentLocation_z_offset), &h_tex_xmachine_message_agentLocation_z_offset, sizeof(int)));
	//bind pbm start and end indices to textures
	size_t tex_xmachine_message_agentLocation_pbm_start_byte_offset;
	size_t tex_xmachine_message_agentLocation_pbm_end_or_count_byte_offset;
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentLocation_pbm_start_byte_offset, tex_xmachine_message_agentLocation_pbm_start, d_agentLocation_partition_matrix->start, sizeof(int)*xmachine_message_agentLocation_grid_size));
	h_tex_xmachine_message_agentLocation_pbm_start_offset = (int)tex_xmachine_message_agentLocation_pbm_start_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentLocation_pbm_start_offset), &h_tex_xmachine_message_agentLocation_pbm_start_offset, sizeof(int)));
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentLocation_pbm_end_or_count_byte_offset, tex_xmachine_message_agentLocation_pbm_end_or_count, d_agentLocation_partition_matrix->end_or_count, sizeof(int)*xmachine_message_agentLocation_grid_size));
  h_tex_xmachine_message_agentLocation_pbm_end_or_count_offset = (int)tex_xmachine_message_agentLocation_pbm_end_or_count_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentLocation_pbm_end_or_count_offset), &h_tex_xmachine_message_agentLocation_pbm_end_or_count_offset, sizeof(int)));

	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	//Set the message_type for non partitioned and spatially partitioned message outputs
	h_message_agentCooperate_output_type = single_message;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_output_type), &h_message_agentCooperate_output_type, sizeof(int)));
	
	
	//MAIN XMACHINE FUNCTION CALL (cooperate)
	//Reallocate   : false
	//Input        : agentLocation
	//Output       : agentCooperate
	//Agent Output : 
	GPUFLAME_cooperate<<<g, b, sm_size, stream>>>(d_persons, d_agentLocations, d_agentLocation_partition_matrix, d_agentCooperates, d_rand48);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//continuous agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentLocation_my_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentLocation_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentLocation_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentLocation_z));
	//unbind pbm indices
    gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentLocation_pbm_start));
    gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentLocation_pbm_end_or_count));
    
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	h_message_agentCooperate_count += h_xmachine_memory_person_count;	
	//Copy count to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_agentCooperate_count), &h_message_agentCooperate_count, sizeof(int)));	
	
#ifdef FAST_ATOMIC_SORTING
  //USE ATOMICS TO BUILD PARTITION BOUNDARY
	//reset partition matrix
	gpuErrchk( hipMemset( (void*) d_agentCooperate_partition_matrix, 0, sizeof(xmachine_message_agentCooperate_PBM)));
  //
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, hist_agentCooperate_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	hist_agentCooperate_messages<<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_agentCooperate_local_bin_index, d_xmachine_message_agentCooperate_unsorted_index, d_agentCooperate_partition_matrix->end_or_count, d_agentCooperates, state_list_size);
	gpuErrchkLaunch();
	
	thrust::device_ptr<int> ptr_count = thrust::device_pointer_cast(d_agentCooperate_partition_matrix->end_or_count);
	thrust::device_ptr<int> ptr_index = thrust::device_pointer_cast(d_agentCooperate_partition_matrix->start);
	thrust::exclusive_scan(thrust::cuda::par.on(stream), ptr_count, ptr_count + xmachine_message_agentCooperate_grid_size, ptr_index); // scan
	
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_agentCooperate_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize; 	// Round up according to array size 
	reorder_agentCooperate_messages <<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_agentCooperate_local_bin_index, d_xmachine_message_agentCooperate_unsorted_index, d_agentCooperate_partition_matrix->start, d_agentCooperates, d_agentCooperates_swap, state_list_size);
	gpuErrchkLaunch();
#else
	//HASH, SORT, REORDER AND BUILD PMB FOR SPATIAL PARTITIONING MESSAGE OUTPUTS
	//Get message hash values for sorting
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, hash_agentCooperate_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	hash_agentCooperate_messages<<<gridSize, blockSize, 0, stream>>>(d_xmachine_message_agentCooperate_keys, d_xmachine_message_agentCooperate_values, d_agentCooperates);
	gpuErrchkLaunch();
	//Sort
	thrust::sort_by_key(thrust::cuda::par.on(stream), thrust::device_pointer_cast(d_xmachine_message_agentCooperate_keys),  thrust::device_pointer_cast(d_xmachine_message_agentCooperate_keys) + h_message_agentCooperate_count,  thrust::device_pointer_cast(d_xmachine_message_agentCooperate_values));
	gpuErrchkLaunch();
	//reorder and build pcb
	gpuErrchk(hipMemset(d_agentCooperate_partition_matrix->start, 0xffffffff, xmachine_message_agentCooperate_grid_size* sizeof(int)));
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_agentCooperate_messages, reorder_messages_sm_size, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	int reorder_sm_size = reorder_messages_sm_size(blockSize);
	reorder_agentCooperate_messages<<<gridSize, blockSize, reorder_sm_size, stream>>>(d_xmachine_message_agentCooperate_keys, d_xmachine_message_agentCooperate_values, d_agentCooperate_partition_matrix, d_agentCooperates, d_agentCooperates_swap);
	gpuErrchkLaunch();
#endif
	//swap ordered list
	xmachine_message_agentCooperate_list* d_agentCooperates_temp = d_agentCooperates;
	d_agentCooperates = d_agentCooperates_swap;
	d_agentCooperates_swap = d_agentCooperates_temp;
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_person_2_count+h_xmachine_memory_person_count > xmachine_memory_person_MAX){
		printf("Error: Buffer size of cooperate agents in state 2 will be exceeded moving working agents to next state in function cooperate\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_person_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_person_Agents<<<gridSize, blockSize, 0, stream>>>(d_persons_2, d_persons, h_xmachine_memory_person_2_count, h_xmachine_memory_person_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_person_2_count += h_xmachine_memory_person_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_2_count), &h_xmachine_memory_person_2_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int person_play_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Continuous agent and message input is spatially partitioned
	sm_size += (blockSize * sizeof(xmachine_message_agentCooperate));
	
	//all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (blockSize * PADDING);
	
	return sm_size;
}

/** person_play
 * Agent function prototype for play function of person agent
 */
void person_play(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_person_2_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_person_2_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_person_list* persons_2_temp = d_persons;
	d_persons = d_persons_2;
	d_persons_2 = persons_2_temp;
	//set working count to current state count
	h_xmachine_memory_person_count = h_xmachine_memory_person_2_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_count), &h_xmachine_memory_person_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_person_2_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_2_count), &h_xmachine_memory_person_2_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_play, person_play_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = person_play_sm_size(blockSize);
	
	
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	//continuous agent with discrete or partitioned message input uses texture caching
	size_t tex_xmachine_message_agentCooperate_source_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_source_id_byte_offset, tex_xmachine_message_agentCooperate_source_id, d_agentCooperates->source_id, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_source_id_offset = (int)tex_xmachine_message_agentCooperate_source_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_source_id_offset), &h_tex_xmachine_message_agentCooperate_source_id_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_destination_id_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_destination_id_byte_offset, tex_xmachine_message_agentCooperate_destination_id, d_agentCooperates->destination_id, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_destination_id_offset = (int)tex_xmachine_message_agentCooperate_destination_id_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_destination_id_offset), &h_tex_xmachine_message_agentCooperate_destination_id_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_cooperate_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_cooperate_byte_offset, tex_xmachine_message_agentCooperate_cooperate, d_agentCooperates->cooperate, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_cooperate_offset = (int)tex_xmachine_message_agentCooperate_cooperate_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_cooperate_offset), &h_tex_xmachine_message_agentCooperate_cooperate_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_x_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_x_byte_offset, tex_xmachine_message_agentCooperate_x, d_agentCooperates->x, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_x_offset = (int)tex_xmachine_message_agentCooperate_x_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_x_offset), &h_tex_xmachine_message_agentCooperate_x_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_y_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_y_byte_offset, tex_xmachine_message_agentCooperate_y, d_agentCooperates->y, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_y_offset = (int)tex_xmachine_message_agentCooperate_y_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_y_offset), &h_tex_xmachine_message_agentCooperate_y_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_z_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_z_byte_offset, tex_xmachine_message_agentCooperate_z, d_agentCooperates->z, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_z_offset = (int)tex_xmachine_message_agentCooperate_z_byte_offset / sizeof(float);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_z_offset), &h_tex_xmachine_message_agentCooperate_z_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message0_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message0_byte_offset, tex_xmachine_message_agentCooperate_message0, d_agentCooperates->message0, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message0_offset = (int)tex_xmachine_message_agentCooperate_message0_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message0_offset), &h_tex_xmachine_message_agentCooperate_message0_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message1_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message1_byte_offset, tex_xmachine_message_agentCooperate_message1, d_agentCooperates->message1, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message1_offset = (int)tex_xmachine_message_agentCooperate_message1_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message1_offset), &h_tex_xmachine_message_agentCooperate_message1_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message2_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message2_byte_offset, tex_xmachine_message_agentCooperate_message2, d_agentCooperates->message2, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message2_offset = (int)tex_xmachine_message_agentCooperate_message2_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message2_offset), &h_tex_xmachine_message_agentCooperate_message2_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message3_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message3_byte_offset, tex_xmachine_message_agentCooperate_message3, d_agentCooperates->message3, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message3_offset = (int)tex_xmachine_message_agentCooperate_message3_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message3_offset), &h_tex_xmachine_message_agentCooperate_message3_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message4_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message4_byte_offset, tex_xmachine_message_agentCooperate_message4, d_agentCooperates->message4, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message4_offset = (int)tex_xmachine_message_agentCooperate_message4_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message4_offset), &h_tex_xmachine_message_agentCooperate_message4_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message5_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message5_byte_offset, tex_xmachine_message_agentCooperate_message5, d_agentCooperates->message5, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message5_offset = (int)tex_xmachine_message_agentCooperate_message5_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message5_offset), &h_tex_xmachine_message_agentCooperate_message5_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message6_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message6_byte_offset, tex_xmachine_message_agentCooperate_message6, d_agentCooperates->message6, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message6_offset = (int)tex_xmachine_message_agentCooperate_message6_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message6_offset), &h_tex_xmachine_message_agentCooperate_message6_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message7_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message7_byte_offset, tex_xmachine_message_agentCooperate_message7, d_agentCooperates->message7, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message7_offset = (int)tex_xmachine_message_agentCooperate_message7_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message7_offset), &h_tex_xmachine_message_agentCooperate_message7_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message8_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message8_byte_offset, tex_xmachine_message_agentCooperate_message8, d_agentCooperates->message8, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message8_offset = (int)tex_xmachine_message_agentCooperate_message8_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message8_offset), &h_tex_xmachine_message_agentCooperate_message8_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message9_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message9_byte_offset, tex_xmachine_message_agentCooperate_message9, d_agentCooperates->message9, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message9_offset = (int)tex_xmachine_message_agentCooperate_message9_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message9_offset), &h_tex_xmachine_message_agentCooperate_message9_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message10_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message10_byte_offset, tex_xmachine_message_agentCooperate_message10, d_agentCooperates->message10, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message10_offset = (int)tex_xmachine_message_agentCooperate_message10_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message10_offset), &h_tex_xmachine_message_agentCooperate_message10_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message11_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message11_byte_offset, tex_xmachine_message_agentCooperate_message11, d_agentCooperates->message11, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message11_offset = (int)tex_xmachine_message_agentCooperate_message11_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message11_offset), &h_tex_xmachine_message_agentCooperate_message11_offset, sizeof(int)));
	size_t tex_xmachine_message_agentCooperate_message12_byte_offset;    
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_message12_byte_offset, tex_xmachine_message_agentCooperate_message12, d_agentCooperates->message12, sizeof(int)*xmachine_message_agentCooperate_MAX));
	h_tex_xmachine_message_agentCooperate_message12_offset = (int)tex_xmachine_message_agentCooperate_message12_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_message12_offset), &h_tex_xmachine_message_agentCooperate_message12_offset, sizeof(int)));
	//bind pbm start and end indices to textures
	size_t tex_xmachine_message_agentCooperate_pbm_start_byte_offset;
	size_t tex_xmachine_message_agentCooperate_pbm_end_or_count_byte_offset;
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_pbm_start_byte_offset, tex_xmachine_message_agentCooperate_pbm_start, d_agentCooperate_partition_matrix->start, sizeof(int)*xmachine_message_agentCooperate_grid_size));
	h_tex_xmachine_message_agentCooperate_pbm_start_offset = (int)tex_xmachine_message_agentCooperate_pbm_start_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_pbm_start_offset), &h_tex_xmachine_message_agentCooperate_pbm_start_offset, sizeof(int)));
	gpuErrchk( hipBindTexture(&tex_xmachine_message_agentCooperate_pbm_end_or_count_byte_offset, tex_xmachine_message_agentCooperate_pbm_end_or_count, d_agentCooperate_partition_matrix->end_or_count, sizeof(int)*xmachine_message_agentCooperate_grid_size));
  h_tex_xmachine_message_agentCooperate_pbm_end_or_count_offset = (int)tex_xmachine_message_agentCooperate_pbm_end_or_count_byte_offset / sizeof(int);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_tex_xmachine_message_agentCooperate_pbm_end_or_count_offset), &h_tex_xmachine_message_agentCooperate_pbm_end_or_count_offset, sizeof(int)));

	
	
	//MAIN XMACHINE FUNCTION CALL (play)
	//Reallocate   : false
	//Input        : agentCooperate
	//Output       : 
	//Agent Output : 
	GPUFLAME_play<<<g, b, sm_size, stream>>>(d_persons, d_agentCooperates, d_agentCooperate_partition_matrix, d_rand48);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	//continuous agent with discrete or partitioned message input uses texture caching
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_source_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_destination_id));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_cooperate));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_x));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_y));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_z));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message0));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message1));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message2));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message3));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message4));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message5));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message6));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message7));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message8));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message9));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message10));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message11));
	gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_message12));
	//unbind pbm indices
    gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_pbm_start));
    gpuErrchk( hipUnbindTexture(tex_xmachine_message_agentCooperate_pbm_end_or_count));
    
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_person_3_count+h_xmachine_memory_person_count > xmachine_memory_person_MAX){
		printf("Error: Buffer size of play agents in state 3 will be exceeded moving working agents to next state in function play\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_person_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_person_Agents<<<gridSize, blockSize, 0, stream>>>(d_persons_3, d_persons, h_xmachine_memory_person_3_count, h_xmachine_memory_person_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_person_3_count += h_xmachine_memory_person_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_3_count), &h_xmachine_memory_person_3_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int person_compute_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** person_compute
 * Agent function prototype for compute function of person agent
 */
void person_compute(hipStream_t &stream){

	int sm_size;
	int blockSize;
	int minGridSize;
	int gridSize;
	int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_person_3_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_person_3_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_person_list* persons_3_temp = d_persons;
	d_persons = d_persons_3;
	d_persons_3 = persons_3_temp;
	//set working count to current state count
	h_xmachine_memory_person_count = h_xmachine_memory_person_3_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_count), &h_xmachine_memory_person_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_person_3_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_3_count), &h_xmachine_memory_person_3_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_compute, person_compute_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = person_compute_sm_size(blockSize);
	
	
	
	
	//MAIN XMACHINE FUNCTION CALL (compute)
	//Reallocate   : false
	//Input        : 
	//Output       : 
	//Agent Output : 
//	GPUFLAME_compute<<<g, b, sm_size, stream>>>(d_persons, d_rand48);
//	gpuErrchkLaunch();
	
//***********************************************************************************************************
        // Start CUDA Visual Profiler
        //hipProfilerStart ();

        // Code inserted to make la TF, JJRG 22-03-2018
        int N = 16384;
        int iterations, resta, numStreams;
        numStreams = 3;
        iterations = agent_number / numStreams;
        resta = agent_number % numStreams;

        //hipfftHandle plan;
        hipfftComplex *data0, *data1, *data2, *data3, *data4, *data5, *data6, *data7;
        hipfftComplex *datadev0, *datadev1, *datadev2, *datadev3, *datadev4, *datadev5, *datadev6, *datadev7;

        if (resta != 0)
             iterations++;
        for (int i = 0; i < iterations; i++)
             {
             //CPU Memory Allocate and initializa
             data0 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             data1 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             data2 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             //data3 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             //data4 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             //data5 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             //data6 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);
             //data7 = (hipfftComplex *) malloc (sizeof (hipfftComplex) * N * 1);

             srand (123456 + i);
             for (int tf = 0; tf < N; tf++)
                  {
                  data0 [tf].x = (float) rand ();
                  data0 [tf].y = (float) rand ();
                  data1 [tf].x = (float) rand ();
                  data1 [tf].y = (float) rand ();
                  data2 [tf].x = (float) rand ();
                  data2 [tf].y = (float) rand ();
                  //data3 [tf].x = (float) rand ();
                  //data3 [tf].y = (float) rand ();
                  //data4 [tf].x = (float) rand ();
                  //data4 [tf].y = (float) rand ();
                  //data5 [tf].x = (float) rand ();
                  //data5 [tf].y = (float) rand ();
                  //data6 [tf].x = (float) rand ();
                  //data6 [tf].y = (float) rand ();
                  //data7 [tf].x = (float) rand ();
                  //data7 [tf].y = (float) rand ();
                  }

             //Registers host memory as page-locked (required for asynch hipMemcpyAsync)
             gpuErrchk (hipHostRegister (data0, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             gpuErrchk (hipHostRegister (data1, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             gpuErrchk (hipHostRegister (data2, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             //gpuErrchk (hipHostRegister (data3, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             //gpuErrchk (hipHostRegister (data4, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             //gpuErrchk (hipHostRegister (data5, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             //gpuErrchk (hipHostRegister (data6, sizeof (hipfftComplex) * N, hipHostRegisterPortable));
             //gpuErrchk (hipHostRegister (data7, sizeof (hipfftComplex) * N, hipHostRegisterPortable));

             //GPU Memory allociate
             gpuErrchk (hipMalloc ((void**) &datadev0, sizeof (hipfftComplex) * N * 1));
             gpuErrchk (hipMalloc ((void**) &datadev1, sizeof (hipfftComplex) * N * 1));
             gpuErrchk (hipMalloc ((void**) &datadev2, sizeof (hipfftComplex) * N * 1));
             //gpuErrchk (hipMalloc ((void**) &datadev3, sizeof (hipfftComplex) * N * 1));
             //gpuErrchk (hipMalloc ((void**) &datadev4, sizeof (hipfftComplex) * N * 1));
             //gpuErrchk (hipMalloc ((void**) &datadev5, sizeof (hipfftComplex) * N * 1));
             //gpuErrchk (hipMalloc ((void**) &datadev6, sizeof (hipfftComplex) * N * 1));
             //gpuErrchk (hipMalloc ((void**) &datadev7, sizeof (hipfftComplex) * N * 1));

             //numStreams = 4;
             printf ("\nAgent number: %d, numStreams: %d\n", agent_number, numStreams);

             //Creates CUDA streams, as many as agent are.
             hipStream_t streams [numStreams];
             for (int i = 0; i < numStreams; i++)
                  gpuErrchk (hipStreamCreate (&streams [i]));

             //Creates cuFFT plans and sets them in streams
             hipfftHandle *plans = (hipfftHandle *) malloc (sizeof (hipfftHandle) * numStreams);
             for (int i = 0; i < numStreams; i++)
                  {
                  hipfftResult res = hipfftPlan1d (&plans [i], N, HIPFFT_C2C, 1);
                  hipfftSetStream (plans [i], streams [i]);
                  }

             //Async memcopyes and computation
             //for (int i = 0; i < numStreams; i++)
             //     {
                  gpuErrchk (hipMemcpyAsync (datadev0, data0, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[0]));
                  gpuErrchk (hipMemcpyAsync (datadev1, data1, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[1]));
                  gpuErrchk (hipMemcpyAsync (datadev2, data2, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[2]));
                  //gpuErrchk (hipMemcpyAsync (datadev3, data3, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[3]));
                  //gpuErrchk (hipMemcpyAsync (datadev4, data4, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[4]));
                  //gpuErrchk (hipMemcpyAsync (datadev5, data5, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[5]));
                  //gpuErrchk (hipMemcpyAsync (datadev6, data6, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[6]));
                  //gpuErrchk (hipMemcpyAsync (datadev7, data7, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice, streams[7]));
                  //gpuErrchk (hipMemcpy (datadev, data, sizeof (hipfftComplex) * N * 1, hipMemcpyHostToDevice));
                  //hipfftResult res = hipfftPlan1d (&plan, N, HIPFFT_C2C, 1);
                  hipfftExecC2C (plans [0], datadev0, datadev0, HIPFFT_FORWARD);
                  hipfftExecC2C (plans [1], datadev1, datadev1, HIPFFT_FORWARD);
                  hipfftExecC2C (plans [2], datadev2, datadev2, HIPFFT_FORWARD);
                  //hipfftExecC2C (plans [3], datadev3, datadev3, HIPFFT_FORWARD);
                  //hipfftExecC2C (plans [4], datadev4, datadev4, HIPFFT_FORWARD);
                  //hipfftExecC2C (plans [5], datadev5, datadev5, HIPFFT_FORWARD);
                  //hipfftExecC2C (plans [6], datadev6, datadev6, HIPFFT_FORWARD);
                  //hipfftExecC2C (plans [7], datadev7, datadev7, HIPFFT_FORWARD);
             //     }

             for (int i = 0; i < numStreams; i++)
                  gpuErrchk (hipStreamSynchronize (streams [i]));
                  //hipDeviceSynchronize ();

             //Releases resources
             gpuErrchk (hipHostUnregister (data0));
             gpuErrchk (hipHostUnregister (data1));
             gpuErrchk (hipHostUnregister (data2));
             //gpuErrchk (hipHostUnregister (data3));
             //gpuErrchk (hipHostUnregister (data4));
             //gpuErrchk (hipHostUnregister (data5));
             //gpuErrchk (hipHostUnregister (data6));
             //gpuErrchk (hipHostUnregister (data7));

             //hipfftDestroy (plan);
             gpuErrchk (hipFree (datadev0));
             gpuErrchk (hipFree (datadev1));
             gpuErrchk (hipFree (datadev2));
             //gpuErrchk (hipFree (datadev3));
             //gpuErrchk (hipFree (datadev4));
             //gpuErrchk (hipFree (datadev5));
             //gpuErrchk (hipFree (datadev6));
             //gpuErrchk (hipFree (datadev7));

             for(int i = 0; i < numStreams; i++)
                  gpuErrchk (hipStreamDestroy (streams [i]));

             free (data0);
             free (data1);
             free (data2);
             //free (data3);
             //free (data4);
             //free (data5);
             //free (data6);
             //free (data7);
             }



        // Stop CUDA Visual Profiler
        //hipProfilerStop ();
//********************************************************************************************************
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_person_start_count+h_xmachine_memory_person_count > xmachine_memory_person_MAX){
		printf("Error: Buffer size of compute agents in state start will be exceeded moving working agents to next state in function compute\n");
		exit(0);
	}
	//append agents to next state list
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_person_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	append_person_Agents<<<gridSize, blockSize, 0, stream>>>(d_persons_start, d_persons, h_xmachine_memory_person_start_count, h_xmachine_memory_person_count);
	gpuErrchkLaunch();
	//update new state agent size
	h_xmachine_memory_person_start_count += h_xmachine_memory_person_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_person_start_count), &h_xmachine_memory_person_start_count, sizeof(int)));	
	
	
}


 
extern "C" void reset_person_start_count()
{
    h_xmachine_memory_person_start_count = 0;
}
 
extern "C" void reset_person_1_count()
{
    h_xmachine_memory_person_1_count = 0;
}
 
extern "C" void reset_person_2_count()
{
    h_xmachine_memory_person_2_count = 0;
}
 
extern "C" void reset_person_3_count()
{
    h_xmachine_memory_person_3_count = 0;
}
